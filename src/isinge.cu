#include "../headers/isinge.cuh"

 void kmcenergy::ising::prepare(float* e_I){

	hipMemcpyToSymbol(HIP_SYMBOL(_energy_I),e_I,sizeof(float) * N_ATOMS_ *N_ATOMS_);

 }

 void kmcenergy::ising::prepare(float* e_I, float r1){

	hipMemcpyToSymbol(HIP_SYMBOL(_energy_I),e_I,sizeof(float) * N_ATOMS_);
	hipMemcpyToSymbol(HIP_SYMBOL(&_r1),&r1,sizeof(float));
 }

 void kmcenergy::ising::prepare(float* e_I, float r1, float r2){

	hipMemcpyToSymbol(HIP_SYMBOL(_energy_I),e_I,sizeof(float) * N_ATOMS_);

	hipMemcpyToSymbol(HIP_SYMBOL(&_r1),&r1,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&_r2),&r2,sizeof(float));
 }


#include "hip/hip_runtime.h"
#include "../headers/kernels.cuh"




#ifndef DEBUG

 __global__ void findNeigboursXyz(const float4 * const sites,
        int4 * neigbours, float3 base1, float3 base2, float3 base3,
        int3 dimensions, float radius, int offset, int beginFrom, int size) {

    int id = blockDim.x * blockIdx.x + threadIdx.x + beginFrom;
    radius = radius*radius;

    extern __shared__ float4 X[];

    int nCntr = 0;
    int i,tile;

    float x = sites[id].x;
    float y = sites[id].y;
    float z = sites[id].z;

    float lx;
    float ly;
    float lz;

    float xp;
    float yp;
    float zp;

    for( i = 0, tile = 0; i < size; i += blockDim.x,++tile)
    {

        if(tile * blockDim.x + threadIdx.x < size)
        {
            float4 d = sites[tile * blockDim.x + threadIdx.x];
            X[threadIdx.x] = d;
        }
        else
        {
            // nie koniecznie, ale lepiej miec pelna kontrole
            X[threadIdx.x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        }

        __syncthreads();

        for (int ii =0; ii < blockDim.x; ++ii)
        {

            if (tile * blockDim.x + ii != id && tile * blockDim.x + ii < size) //<<---- (ii != id jest OK)
            {
                xp = X[ii].x - x;
                yp = X[ii].y - y;
                zp = X[ii].z - z;

                #pragma unroll
                for (int k = -1; k <= 1; ++k)
                {
                    #pragma unroll
                    for (int l = -1; l <= 1; ++l)
                    {
                        #pragma unroll
                        for (int m = -1; m <= 1; ++m)
                        {
                            lx = k * dimensions.x * base1.x
                                    + l * dimensions.y * base2.x
                                    + m * dimensions.z * base3.x;
                            ly = k * dimensions.x * base1.y
                                    + l * dimensions.y * base2.y
                                    + m * dimensions.z * base3.y;
                            lz = k * dimensions.x * base1.z
                                    + l * dimensions.y * base2.z
                                    + m * dimensions.z * base3.z;

                            float distance = (
                                    (xp + lx) * (xp + lx)
                                    + (yp + ly) * (yp + ly)
                                    + (zp + lz) * (zp + lz));

                            if (distance < radius && nCntr < offset)
                            {
                                neigbours[id * offset + nCntr].x = -k;
                                neigbours[id * offset + nCntr].y = -l;
                                neigbours[id * offset + nCntr].z = -m;
                                neigbours[id * offset + nCntr].w =  tile * blockDim.x + ii;
                                nCntr++;
                            }
                        }
                    }
                }
            }
        }

        __syncthreads(); //<<---- bez tego nie ma kontroli nad shared bufforem X.
    }
}

#endif

 /*
  * set float4 fields
  */


 __global__ void setFloat4x(int index,float value,float4* input, int size) {
  	int thId = blockIdx.x * blockDim.x + threadIdx.x;
  	if(thId < size && index == thId)
  		input[thId].x = value;
  }

 __global__ void setFloat4y(int index,float value,float4* input, int size) {
  	int thId = blockIdx.x * blockDim.x + threadIdx.x;
  	if(thId < size && index == thId)
  		input[thId].y = value;
  }

 __global__ void setFloat4z(int index,float value,float4* input, int size) {
  	int thId = blockIdx.x * blockDim.x + threadIdx.x;
  	if(thId < size && index == thId)
  		input[thId].z = value;
  }
 __global__ void setFloat4w(int index,float value,float4* input, int size) {
 	int thId = blockIdx.x * blockDim.x + threadIdx.x;
 	if(thId < size && index == thId)
 		input[thId].w = value;
 }


 /*
  * Exchange values in Float4 fields
  */

 __global__ void  exchangeFloat4x(int index1, int index2,float4* input, int size) {
 	int thId = blockIdx.x * blockDim.x + threadIdx.x;
 	if(thId < size && thId == index1) {
 		float temp = input[thId].x;
 		input[thId].x = input[index2].x;
 		input[index2].x = temp;
 	}
 }

 __global__ void  exchangeFloat4y(int index1, int index2,float4* input, int size) {
 	int thId = blockIdx.x * blockDim.x + threadIdx.x;
 	if(thId < size && thId == index1) {
 		float temp = input[thId].y;
 		input[thId].y = input[index2].y;
 		input[index2].y = temp;
 	}
 }

 __global__ void  exchangeFloat4z(int index1, int index2,float4* input, int size) {
 	int thId = blockIdx.x * blockDim.x + threadIdx.x;
 	if(thId < size && thId == index1) {
 		float temp = input[thId].z;
 		input[thId].z = input[index2].z;
 		input[index2].z = temp;
 	}
 }

 __global__ void  exchangeFloat4w(int index1, int index2,float4* input, int size) {
 	int thId = blockIdx.x * blockDim.x + threadIdx.x;

 	if(thId < size && thId == index1) {
 		float temp = input[thId].w;
 		input[thId].w = input[index2].w;
 		input[index2].w = temp;
 	}
 }






 /* To jest wersja działająca - nie jest w inej używana pamięć shared*/


#ifdef DEBUG

 __global__ void findNeigboursXyz(const float4 * const sites,
		int4 * neigbours, float3 base1, float3 base2, float3 base3,
		int3 dimensions, float radius, int offset, int beginFrom, int size) {

	int id = blockDim.x * blockIdx.x + threadIdx.x + beginFrom;

	if (id < size) {

		float x = sites[id].x;
		float y = sites[id].y;
		float z = sites[id].z;

		float lx;
		float ly;
		float lz;

		int nCntr = 0;

		for (int i = beginFrom; i < size; ++i) {
			if (i != id) {

				float xp = sites[i].x - x;
				float yp = sites[i].y - y;
				float zp = sites[i].z - z;

#pragma unroll
				for (int k = -1; k <= 1; ++k) {
#pragma unroll
					for (int l = -1; l <= 1; ++l) {
#pragma unroll
						for (int m = -1; m <= 1; ++m) {

							lx = k * dimensions.x * base1.x
									+ l * dimensions.y * base2.x
									+ m * dimensions.z * base3.x;
							ly = k * dimensions.x * base1.y
									+ l * dimensions.y * base2.y
									+ m * dimensions.z * base3.y;
							lz = k * dimensions.x * base1.z
									+ l * dimensions.y * base2.z
									+ m * dimensions.z * base3.z;

							float distance = sqrt(
									(xp + lx) * (xp + lx)
									+ (yp + ly) * (yp + ly)
									+ (zp + lz) * (zp + lz));

							if (distance < radius && nCntr < offset) {

								neigbours[id * offset + nCntr].x = -k;
								neigbours[id * offset + nCntr].y = -l;
								neigbours[id * offset + nCntr].z = -m;
								neigbours[id * offset + nCntr].w =  i;
								nCntr++;
							}
						}
					}
				}
			}
		}

	}
}

#endif

